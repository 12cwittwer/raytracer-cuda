#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "raytracer/vec3.h"
#include "raytracer/ray.h"
#include "raytracer/interval.h"
#include "raytracer/color.h"

#include "raytracer/camera_data.h"
#include "raytracer/material.h"
#include "raytracer/hittable.h"
#include "raytracer/hittable_dispatch.h"
#include "raytracer/hittable_dispatch_impl.h"
#include "raytracer/bvh.h"
#include "raytracer/sphere_gpu.h"
#include "raytracer/quad.h"
#include "raytracer/instances.h"
#include "raytracer/device_ray_color.h"
#include "raytracer/random_utils.h"

#include "raytracer/render.h"


__global__ void render_kernel(
    const camera_data* cam,
    const hittable* world,
    color* framebuffer,
    int row
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = row;

    if (x >= cam->image_width) return;

    int pixel_index = x;
    hiprandState rng;
    int seed = 1984 + row * cam->image_width + x;
    hiprand_init(seed, 0, 0, &rng);

    color pixel_color(0, 0, 0);
    for (int s = 0; s < cam->samples_per_pixel; ++s) {
        ray r = get_ray(cam, x, y, &rng);
        pixel_color += ray_color(r, cam->max_depth, world, cam->background, rng);
    }
    pixel_color /= cam->samples_per_pixel;
    framebuffer[pixel_index] = pixel_color;
}

#include "raytracer/cuda_utils.h"  // for CUDA_CHECK

void launch_render_kernel(const camera_data* cam, const hittable* world, color* fb, int image_width, int image_height, int row) {
    const dim3 threads_per_block(512, 1);
    const dim3 num_blocks((image_width + threads_per_block.x - 1) / threads_per_block.x, 1);

    // Launch kernel
    // Try rendering the first pixel
    // render_kernel<<<1, 1>>>(cam, world, fb);
    render_kernel<<<num_blocks, threads_per_block>>>(cam, world, fb, row);

    // Check for immediate kernel launch errors
    CUDA_CHECK(hipGetLastError());

    // Ensure kernel is finished before moving on
    // CUDA_CHECK(hipDeviceSynchronize());
}
