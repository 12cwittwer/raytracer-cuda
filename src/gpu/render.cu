#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "raytracer/vec3.h"
#include "raytracer/ray.h"
#include "raytracer/interval.h"
#include "raytracer/color.h"

#include "raytracer/camera_data.h"
#include "raytracer/material.h"
#include "raytracer/hittable.h"
#include "raytracer/hittable_dispatch.h"
#include "raytracer/hittable_dispatch_impl.h"
#include "raytracer/bvh.h"
#include "raytracer/sphere_gpu.h"
#include "raytracer/quad.h"
#include "raytracer/instances.h"
#include "raytracer/device_ray_color.h"
#include "raytracer/random_utils.h"

#include "raytracer/render.h"


__global__ void render_kernel(
    const camera_data* cam,
    const hittable* world,
    color* framebuffer
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x == 0 && y == 0) {
        printf("Kernel launched\n");
    }

    if (x >= cam->image_width || y >= cam->image_height) return;

    if (x == 0 && y == 0) {
        printf("Camera Dimensions Width: %d Height: %d\n", cam->image_width, cam->image_height);
    }

    if (x == 0 && y == 0) {
        printf("First hittable is %d\n", (int)world->type);
    }

    int pixel_index = y * cam->image_width + x;
    hiprandState rng;
    hiprand_init(1984 + pixel_index, 0, 0, &rng);

    color pixel_color(0, 0, 0);
    for (int s = 0; s < cam->samples_per_pixel; ++s) {
        ray r = get_ray(cam, x, y, &rng);
        pixel_color += ray_color(r, cam->max_depth, world, cam->background, rng);
    }

    framebuffer[pixel_index] = pixel_color / cam->samples_per_pixel;
}

void launch_render_kernel(const camera_data* cam, const hittable* world, color* fb, int image_width, int image_height) {
    dim3 threads_per_block(8, 8);
    dim3 num_blocks(
        (image_width + threads_per_block.x - 1) / threads_per_block.x,
        (image_height + threads_per_block.y - 1) / threads_per_block.y
    );
    render_kernel<<<num_blocks, threads_per_block>>>(cam, world, fb);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Launch Error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();
}
