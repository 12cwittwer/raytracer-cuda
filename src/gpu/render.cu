#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "raytracer/vec3.h"
#include "raytracer/ray.h"
#include "raytracer/interval.h"
#include "raytracer/color.h"

#include "raytracer/camera_data.h"
#include "raytracer/material.h"
#include "raytracer/hittable.h"
#include "raytracer/hittable_dispatch.h"
#include "raytracer/hittable_dispatch_impl.h"
#include "raytracer/bvh.h"
#include "raytracer/sphere_gpu.h"
#include "raytracer/quad.h"
#include "raytracer/instances.h"
#include "raytracer/device_ray_color.h"
#include "raytracer/random_utils.h"

#include "raytracer/render.h"


__global__ void render_kernel(
    const camera_data* cam,
    const hittable* world,
    color* framebuffer
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x == 0 && y == 0) {
        printf("Kernel launched\n");
    }

    if (x >= cam->image_width || y >= cam->image_height) return;

    if (x == 0 && y == 0) {
        printf("Camera Dimensions Width: %d Height: %d\n", cam->image_width, cam->image_height);
    }

    if (x == 0 && y == 0) {
        printf("First hittable is %d\n", (int)world->type);
        printf("World data is %p\n", world->data);
    }

    printf("World first hittable is %d\n", (int)world->type);
    printf("World data is %p\n", world->data);

    int pixel_index = y * cam->image_width + x;
    hiprandState rng;
    hiprand_init(1984 + pixel_index, 0, 0, &rng);

    color pixel_color(0, 0, 0);
    for (int s = 0; s < cam->samples_per_pixel; ++s) {
        if (x == 0 && y == 0) {
            printf("cam=%p\n", cam);
            printf("cam->image_width=%d\n", cam->image_width);
            printf("cam->samples_per_pixel=%d\n", cam->samples_per_pixel);
            printf("cam->background=(%f, %f, %f)\n",
                cam->background.x(), cam->background.y(), cam->background.z());
        }
        ray r = get_ray(cam, x, y, &rng);
        if (world->data == nullptr) {
            if (x == 0 && y == 0) printf("CRASH PREVENTED: world->data was null\n");
            return;
        }
        pixel_color += ray_color(r, cam->max_depth, world, cam->background, rng);
    }

    framebuffer[pixel_index] = pixel_color / cam->samples_per_pixel;
}

#include "raytracer/cuda_utils.h"  // for CUDA_CHECK

void launch_render_kernel(const camera_data* cam, const hittable* world, color* fb, int image_width, int image_height) {
    const dim3 threads_per_block(8, 8);
    const dim3 num_blocks(
        (image_width + threads_per_block.x - 1) / threads_per_block.x,
        (image_height + threads_per_block.y - 1) / threads_per_block.y
    );

    // Launch kernel
    render_kernel<<<num_blocks, threads_per_block>>>(cam, world, fb);

    // Check for immediate kernel launch errors
    CUDA_CHECK(hipGetLastError());

    // Ensure kernel is finished before moving on
    CUDA_CHECK(hipDeviceSynchronize());
}
